
#include <hip/hip_runtime.h>
namespace br { namespace cuda { namespace cudacopyto {
  template <typename T> void wrapper(const T* in, void** out, const int rows, const int cols) {
    hipMalloc(out, rows*cols*sizeof(T));
    hipMemcpy(*out, in, rows*cols*sizeof(T), hipMemcpyHostToDevice);
  }

  template void wrapper(const float* in, void** out, const int rows, const int cols);
  template void wrapper(const unsigned char* in, void** out, const int rows, const int cols);
}}}
