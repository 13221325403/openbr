
#include <hip/hip_runtime.h>
namespace br { namespace cuda { namespace cudacopyto {
  //template<typename T>
  //void wrapper(const T* in, void** out, const int rows, const int cols) {
  void wrapper(const unsigned char* in, void** out, const int rows, const int cols) {
    hipMalloc(out, rows*cols*sizeof(unsigned char));
    hipMemcpy(*out, in, rows*cols*sizeof(unsigned char), hipMemcpyHostToDevice);
  }
}}}
