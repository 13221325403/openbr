#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;

#include <opencv2/opencv.hpp>
#include <opencv2/gpu/gpu.hpp>

#include "cudadefines.hpp"

using namespace cv;
using namespace cv::gpu;

namespace br { namespace cuda { namespace pca {
  __global__ void multiplyKernel(float* src, float* dst, float* evPtr, int evRows, int evCols) {
    int colInd = blockIdx.x*blockDim.x+threadIdx.x;

    // check dimensions
    if (colInd >= evCols) {
      return;
    }

    dst[colInd] = 0;
    for (int i=0; i < evRows; i++) {
      dst[colInd] += evPtr[evCols*i + colInd] * src[i];
    }
  }

  __global__ void subtractMeanKernel(float* out, float* mean, int numCols) {
    int colInd = blockIdx.x*blockDim.x+threadIdx.x;

    // perform bound checking
    if (colInd >= numCols) {
      return;
    }

    // subtract out the mean
    out[colInd] -= mean[colInd];
  }

  float* cudaEvPtr; int _evRows; int _evCols;
  float* cudaMeanPtr; int _meanElems;
  float* _cudaSrcPtr;
  float* _cudaDstPtr;

  void loadwrapper(float* evPtr, int evRows, int evCols, float* meanPtr, int meanElems) {
    _evRows = evRows; _evCols = evCols;
    _meanElems = meanElems;

    hipError_t err;

    // copy the eigenvectors to the GPU
    CUDA_SAFE_MALLOC(&cudaEvPtr, evRows*evCols*sizeof(float), &err);
    CUDA_SAFE_MEMCPY(cudaEvPtr, evPtr, evRows*evCols*sizeof(float), hipMemcpyHostToDevice, &err);

    // copy the mean to the GPU
    CUDA_SAFE_MALLOC(&cudaMeanPtr, meanElems*sizeof(float), &err);
    CUDA_SAFE_MEMCPY(cudaMeanPtr, meanPtr, meanElems*sizeof(float), hipMemcpyHostToDevice, &err);

    CUDA_SAFE_MALLOC(&_cudaSrcPtr, _meanElems*sizeof(float), &err);
    CUDA_SAFE_MALLOC(&_cudaDstPtr, _evCols*sizeof(float), &err);
  }

  void wrapper(void* src, void** dst) {
    // copy the image to the GPU
    //hipMemcpy(_cudaSrcPtr, src, _meanElems*sizeof(float), hipMemcpyHostToDevice);
    hipError_t err;
    CUDA_SAFE_MALLOC(dst, _evRows*_evCols*sizeof(float), &err);

    // subtract out the mean of the image (mean is 1xpixels in size)
    int threadsPerBlock = 64;
    int numBlocks = _meanElems / threadsPerBlock + 1;
    subtractMeanKernel<<<numBlocks, threadsPerBlock>>>((float*)src, cudaMeanPtr, _meanElems);
    CUDA_KERNEL_ERR_CHK(&err);

    // perform the multiplication
    threadsPerBlock = 64;
    numBlocks = _evCols / threadsPerBlock + 1;
    multiplyKernel<<<numBlocks, threadsPerBlock>>>((float*)src, (float*)(*dst), cudaEvPtr, _evRows, _evCols);
    CUDA_KERNEL_ERR_CHK(&err);

    CUDA_SAFE_FREE(src, &err);    // TODO(colin): figure out why adding this free causes memory corruption...

    // copy the data back to the CPU
    //hipMemcpy(dst, _cudaDstPtr, _evCols*sizeof(float), hipMemcpyDeviceToHost);
  }
}}}
